#include "hip/hip_runtime.h"
// PLEASE MODIFY THIS FILE TO IMPLEMENT YOUR SOLUTION

#include "apsp.h"
#include "stdio.h"
constexpr int b = 32;
constexpr int dmax = 100000 + 1; // edge weight < 100000 is guaranteed

namespace
{

__global__ void step1(const int p, const int n, int* graph)
{
    // handles the p-th diagnal block

    // copy the p-th block to shared mem first!
    __shared__ int cache[b][b];
    // coordinate order:
    //         x
    //       <--->
    //    A
    //  y |  (blk)
    //    V

    // local coordinates, e.g. cache[y][x]
    const int x = threadIdx.x;
    const int y = threadIdx.y;
    // global coordinates (on graph), e.g. graph[yg*n + xg]
    const int xg = p * blockDim.x + x;
    const int yg = p * blockDim.y + y;

    if (xg < n && yg < n)
    {
        cache[y][x] = graph[yg * n + xg];
    }
    else
    {
        cache[y][x] = dmax;
    }

    __syncthreads(); // wait till all data prepared at shared mem

    // all blks will act like diagnal blks (no exp time wasted...)
    int newchoice;
    #pragma unroll
    for (int k = 0; k < b; k++)
    {
        newchoice = cache[y][k] + cache[k][x];
        __syncthreads(); // cache will be modified in the following lines
        cache[y][x] = min(cache[y][x], newchoice);
        __syncthreads();
    }

    // send results back to global memory
    if (xg < n && yg < n)
    {
        graph[yg * n + xg] = cache[y][x];
    }

}

__global__ void step2(const int p, const int n, int* graph)
{
    // handles the path between the p-th diagnal block and other blocks
    // only a cross-shape block field is related to the p-th block
    if (blockIdx.x == p)
    {
        return;     // could be better?
    }

    const int x = threadIdx.x;
    const int y = threadIdx.y;

    // global coordnate of p-th diag block
    int xg = p * blockDim.x + x;
    int yg = p * blockDim.y + y;

    // printf("p=%d,x=%d,y=%d,xg=%d,yg=%d\n",p,x,y,xg,yg);

    // have to load 2 blks to shared memory
    // 1st: the p-th diagnal block
    // 2nd: the corresponding 'other block' determined by blkIdx

    __shared__ int diagnal[b][b];
    __shared__ int cache[b][b];     // self block

    // 1st
    if (xg < n && yg < n)
    {
        diagnal[y][x] = graph[yg * n + xg];
    }
    else
    {
        diagnal[y][x] = dmax;
    }

    // 2nd
    if (blockIdx.y == 0)
    {   // redirect xg,yg to current block to save space
        xg = blockDim.x * blockIdx.x + x; // row
        
    }
    else
    {
        yg = blockDim.x * blockIdx.x + y; // col
    }
    if (xg < n && yg < n)
    {
        cache[y][x] = graph[yg * n + xg];
    }
    else
    {
        cache[y][x] = dmax;
    }

    __syncthreads();

    // then update cache
    int newchoice;
    if (blockIdx.y == 0) // row
    {
        #pragma unroll
        for (int k = 0; k < blockDim.x; k++)
        {
            newchoice = diagnal[y][k] + cache[k][x];
            __syncthreads();
            cache[y][x] = min(cache[y][x], newchoice);
            __syncthreads();
        }
    }
    else // col
    {
        #pragma unroll
        for (int k = 0; k < blockDim.x; k++)
        {
            newchoice = cache[y][k] + diagnal[k][x];
            __syncthreads();
            cache[y][x] = min(cache[y][x], newchoice);
            __syncthreads();
        }
    }

    // send results back to global memory
    // xg,yg refers to current block so directly use it
    if (xg < n && yg < n)
    {
        graph[yg * n + xg] = cache[y][x];
    }
}

__global__ void step3(const int p, const int n, int* graph)
{
    // handles the rest blocks
    // quit if current block is in the 'cross field'
    if (blockIdx.x == p || blockIdx.y == p)
    {
        return;
    }

    // local coordinates, e.g. cache[y][x]
    const int x = threadIdx.x;
    const int y = threadIdx.y;
    // global coordinates (on graph), e.g. graph[yg*n + xg]
    const int xg = blockIdx.x * blockDim.x + x;
    const int yg = blockIdx.y * blockDim.y + y;
    const int xgp = p * blockDim.x + x;
    const int ygp = p * blockDim.y + y;

    __shared__ int rowblk[b][b];    // in the same row
    __shared__ int colblk[b][b];    // in the same col

    // load 2 blocks to shared memory
    if (xgp < n && yg < n)
    {
        rowblk[y][x] = graph[yg * n + xgp];
    }
    else
    {
        rowblk[y][x] = dmax;
    }
    if (xg < n && ygp < n)
    {
        colblk[y][x] = graph[ygp * n + xg];
    }
    else
    {
        colblk[y][x] = dmax;
    }
    __syncthreads();

    // update!
    int newchoice;
    if (xg < n && yg < n)
    {
	newchoice = graph[yg * n + xg];
    	#pragma unroll
    	for (int k = 0; k < b; k++)
    	{
            newchoice = min(newchoice, rowblk[y][k] + colblk[k][x]);
    	}
    	// send results back to global memory
        graph[yg * n + xg] = newchoice;
    }
}


}   //namespace

void printcudamem(/* device */ int *graph, int n)
{
    int* tmp = (int*)malloc(n*n);
    hipMemcpy(tmp, graph, n*n, hipMemcpyDeviceToHost);
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%d ",tmp[i*n+j]);
        }
        printf("\n");
    }

    free(tmp);
}

void apsp(int n, /* device */ int *graph) {
    dim3 thr(b, b);     // size of each block
    dim3 nblk_s1(1,1);                               // diagnal
    dim3 nblk_s2((n - 1) / b + 1,2);                 // cross
    // .y=0-> row, .y=1-> col, .x-> the order in row or col
    dim3 nblk_s3((n - 1) / b + 1,(n - 1) / b + 1);   // all
    for (int p = 0; p < (n - 1) / b + 1; p++)
    {
        step1<<<nblk_s1, thr>>>(p, n, graph);
        step2<<<nblk_s2, thr>>>(p, n, graph);
        step3<<<nblk_s3, thr>>>(p, n, graph);
    }
}

