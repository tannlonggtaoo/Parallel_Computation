#include "hip/hip_runtime.h"
// reference : GE-SpMM (arXiv:2007.03179)
#include "spmm_opt.h"

constexpr int BLOCK_SIZE = 32;
constexpr int WARP_SIZE = 32;

__global__ void spmm_kernel_placeholder(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int INFEATURE)
{	
	const int rid = blockIdx.x * BLOCK_SIZE + threadIdx.y;
	// each thread block handles BLOCK_SIZE rows
	const int cid = blockIdx.y * WARP_SIZE + threadIdx.x;
	// 0 to feat_in
    const int tid = threadIdx.y * WARP_SIZE + threadIdx.x;	
	// 0 to WARP_SIZE*BLOCK_SIZE
	if (rid >= num_v) return;

	extern __shared__ int sm[];
	int* sm_k = sm;   									// for caching idx
	float* sm_v = (float*)sm + BLOCK_SIZE * WARP_SIZE;  // for caching val
	int sm_base = threadIdx.y * WARP_SIZE;				// 0,32,64,...

	int begin = ptr[rid], end = ptr[rid + 1];
	float result = 0.0f;
	int k;

	// iteration over whole row
	for (int p = begin; p < end; p+=WARP_SIZE)
	{
		// loading A (caching)
		if (p + threadIdx.x < end)
		{
			sm_k[tid] = idx[p + threadIdx.x];
			sm_v[tid] = val[p + threadIdx.x];
		}
		__syncwarp();	// wait till all float num loaded
		
		// computation
		for (int kk = 0; kk < WARP_SIZE; kk++)
		{
			if (p + kk < end)
			{
				k = sm_k[sm_base + kk];		// corresponding idx
				if (cid < INFEATURE)
				{
					result += sm_v[sm_base + kk] * vin[k * INFEATURE + cid];
				}
			}
		}
		__syncwarp();
	}
	if (cid < INFEATURE)
	{
		vout[rid * INFEATURE + cid] = result;
	}
}
void SpMMOpt::preprocess(float *vin, float *vout)
{
    // TODO: your code
    grid.x = (num_v + BLOCK_SIZE - 1) / BLOCK_SIZE;
	grid.y = (feat_in + WARP_SIZE - 1) / WARP_SIZE;
    block.x = WARP_SIZE;
	block.y = BLOCK_SIZE;
}

void SpMMOpt::run(float *vin, float *vout)
{
    // TODO: your code
    spmm_kernel_placeholder<<<grid, block, WARP_SIZE*BLOCK_SIZE*(sizeof(int) + sizeof(float))>>>(d_ptr, d_idx, d_val, vin, vout, num_v, feat_in);
}
