#include "hip/hip_runtime.h"
#include "spmm_opt.h"

constexpr int THRBLK_SIZE = 128; // thread block size
constexpr int ROWBLK_SIZE = 32;  // #rows for each thread block
constexpr int INFEATURE_MAX = 256

__global__ void spmm_kernel_placeholder(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int INFEATURE)
{
    // STEP 1 preparation
    // params for the whole thread block
    const int rblklo = blockIdx.x * ROWBLK_SIZE; // row block begin row
    const int rblkhi = min((blockIdx.x + 1) * blockDim.x, num_v); // row block end row (not included)
    const int valcnt = (ptr[rblkhi] - ptr[rblklo] + THRBLK_SIZE - 1) / THRBLK_SIZE; // len(ptr) = num_v + 1
    __shared__ int ansbuf[ROWBLK_SIZE][INFEATURE_MAX]; // should use constexpr so define INFEATURE_MAX (assume INFEATURE <= 256)
    // shared memory is NOT INITIALIZED
    int cnt4eachthr = ROWBLK_SIZE * INFEATURE / THRBLK_SIZE;
    for (int i = cnt4eachthr * threadIdx.x; i < cnt4eachthr * (threadIdx.x + 1); i++)
    {
        ((int*)ansbuf)[i] = 0;
    }
    __syncthreads();
    // (segmentation here is the same as STEP 3)

    // params for this thread
    const int vallo = ptr[rblklo] + threadIdx.x * valcnt;
    const int valhi = min(ptr[rblklo] + (threadIdx.x + 1) * valcnt, ptr[rblkhi]) // not included as well
    
    // find which row(s) this thread is handling
    int rlo = rblklo;
    while (ptr[rlo] <= vallo) rlo++;
    rlo--;
    int rhi = rlo;
    while (ptr[rhi] <= valhi) rhi++; // rhi not included

    // STEP 2 computation
    // all atomic
    for (int r = rlo; r < rhi; r++)
    {
        // low efficiency
        int ibegin = (r == rlo) ? vallo : ptr[r];
        int iend = (r == (rhi - 1)) ? valhi : ptr[r+1];
        for (int j = 0; j < INFEATURE; j++)
        {
            // j: col index of B
            float result = 0.0f;
            for (int i = ibegin; i < iend; i++)
            {
                result += vin[idx[i] * INFEATURE + j] * val[i];
            }
            // may try other scopes (now on device)
            atomicAdd(&(ansbuf[r][j]), result);
        }
    }
    __syncthreads();

    // STEP 3 move to global memory
    for (int i = cnt4eachthr * threadIdx.x; i < cnt4eachthr * (threadIdx.x + 1); i++)
    {
        vout[rblklo * INFEATURE + i] = ((int*)ansbuf)[i];
    }
}
void SpMMOpt::preprocess(float *vin, float *vout)
{
    grid.x = (num_v + ROWBLK_SIZE - 1) / ROWBLK_SIZE; // num_v = #rows in total
    block.x = THRBLK_SIZE;
    // equalization within each thread block (ROWBLK_SIZE rows)...

}

void SpMMOpt::run(float *vin, float *vout)
{
    // TODO: your code
    spmm_kernel_placeholder<<<grid, block>>>(d_ptr, d_idx, d_val, vin, vout, num_v, feat_in);
}