#include "hip/hip_runtime.h"
#include "spmm_opt.h"

__global__ void spmm_kernel_placeholder(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int INFEATURE)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_v) return;
    int begin = ptr[tid], end = ptr[tid + 1];
    for (int j = 0; j < INFEATURE; ++j)
    {
        float result = 0.0f;
        for (int i = begin; i < end; ++i)
        {
            result += vin[idx[i] * INFEATURE + j] * val[i];
        }
        vout[tid * INFEATURE + j] = result;
    }
}
void SpMMOpt::preprocess(float *vin, float *vout)
{
    // TODO: your code
    int BLOCK_SIZE = 128;
    grid.x = (num_v + BLOCK_SIZE - 1) / BLOCK_SIZE;
    block.x = BLOCK_SIZE;
}

void SpMMOpt::run(float *vin, float *vout)
{
    // TODO: your code
    spmm_kernel_placeholder<<<grid, block>>>(d_ptr, d_idx, d_val, vin, vout, num_v, feat_in);
}