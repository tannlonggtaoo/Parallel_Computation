#include "hip/hip_runtime.h"
// reference : GE-SpMM (arXiv:2007.03179)
#include "spmm_opt.h"

constexpr int BLOCK_SIZE = 32;
constexpr int WARP_SIZE = 32;

__global__ void spmm_kernel_placeholder(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int INFEATURE)
{	
	const int rid = blockIdx.x * BLOCK_SIZE + threadIdx.y;
	// each thread block handles BLOCK_SIZE rows
	const int cid = blockIdx.y * WARP_SIZE + threadIdx.x;
	// 0 to feat_in
    const int tid = threadIdx.y * WARP_SIZE + threadIdx.x;	
	// 0 to WARP_SIZE*BLOCK_SIZE
	if ((rid >= num_v) || (cid >= INFEATURE)) return;

	extern __shared__ int sm[];
	int* sm_k = sm;   									// for caching idx
	float* sm_v = (float*)sm + BLOCK_SIZE * WARP_SIZE;  // for caching val
	int sm_base = threadIdx.y * WARP_SIZE;				// 0,32,64,...

	int begin = ptr[rid], end = ptr[rid + 1];
	float result = 0.0f;
	int k,p,kk,cur;		// iter vars

	// iteration over whole row
	for (p = begin; p < end; p+=WARP_SIZE)
	{
		cur = p + threadIdx.x;
		// loading A (caching)
		if (cur < end)
		{
			sm_k[tid] = idx[cur];
			sm_v[tid] = val[cur];
		}
		__syncwarp();	// wait till all float num loaded
		
		// computation
		for (kk = 0; (kk < WARP_SIZE) && (p + kk < end); kk++)
		{
			k = sm_k[sm_base + kk];		// corresponding idx
			result += sm_v[sm_base + kk] * vin[k * INFEATURE + cid];
		}
		__syncwarp();
	}
	vout[rid * INFEATURE + cid] = result;
}
void SpMMOpt::preprocess(float *vin, float *vout)
{
    // TODO: your code
    grid.x = (num_v + BLOCK_SIZE - 1) / BLOCK_SIZE;
	grid.y = (feat_in + WARP_SIZE - 1) / WARP_SIZE;
    block.x = WARP_SIZE;
	block.y = BLOCK_SIZE;
}

void SpMMOpt::run(float *vin, float *vout)
{
    // TODO: your code
    spmm_kernel_placeholder<<<grid, block, WARP_SIZE*BLOCK_SIZE*(sizeof(int) + sizeof(float))>>>(d_ptr, d_idx, d_val, vin, vout, num_v, feat_in);
}
