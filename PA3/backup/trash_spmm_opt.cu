#include "hip/hip_runtime.h"
#include "spmm_opt.h"

constexpr int THRBLK_SIZE = 8; // thread block size
constexpr int ROWBLK_SIZE = 4;  // #rows for each thread block
constexpr int INFEATURE_MAX = 1;
//constexpr int THRBLK_SIZE = 128; // thread block size
//constexpr int ROWBLK_SIZE = 48;  // #rows for each thread block
//constexpr int INFEATURE_MAX = 256;



__global__ void spmm_kernel_placeholder(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int INFEATURE)
{
	// vin is B !!!


	//debug	
	if (blockIdx.x==0 && threadIdx.x == 0) printf("num_v=%d,INFEATURE=%d\n",num_v,INFEATURE);
    
	// STEP 1 preparation
    // params for the whole thread block
    const int rblklo = blockIdx.x * ROWBLK_SIZE; // row block begin row
    const int rblkhi = min((blockIdx.x + 1) * ROWBLK_SIZE, num_v); // row block end row (not included)

    const int valcnt = (ptr[rblkhi] - ptr[rblklo] + THRBLK_SIZE - 1) / THRBLK_SIZE; // len(ptr) = num_v + 1
    __shared__ float ansbuf[ROWBLK_SIZE * INFEATURE_MAX]; // should use constexpr so define INFEATURE_MAX (assume INFEATURE <= 256)
    // shared memory is NOT INITIALIZED
    int cnt4eachthr = ROWBLK_SIZE * INFEATURE_MAX / THRBLK_SIZE;

    // debug = 8
    //if (blockIdx.x==17) printf("cnt4eachthr%d\n",cnt4eachthr);
    //return;

    for (int i = cnt4eachthr * threadIdx.x; i < cnt4eachthr * (threadIdx.x + 1); i++)
    {
        ansbuf[i] = 0.0f;

		// debug
		// if(blockIdx.x==0) printf("threadidx=%d,ansbuf[%d][%d]=0\n",threadIdx.x,i/INFEATURE_MAX,i%INFEATURE_MAX);
    }
    __syncthreads();
    // (segmentation here is the same as STEP 3)

    // params for this thread

    // debug
    //if(blockIdx.x==0) printf("blockIdx=%d,threadIdx=%d,blockDim=%d,rblklo=%d,rblkhi=%d,valcnt=%d\n",blockIdx.x,threadIdx.x,blockDim.x,rblklo,rblkhi,valcnt);
    // for debugging (at least OK here...)

    const int vallo = ptr[rblklo] + threadIdx.x * valcnt;
    const int valhi = min(ptr[rblklo] + (threadIdx.x + 1) * valcnt, ptr[rblkhi]); // not included as well

	// debug
	// if(blockIdx.x==0) printf("threadIdx=%d,vallo=%d,valhi=%d,totalnum4blk=%d\n",threadIdx.x,vallo,valhi,ptr[rblkhi]-ptr[rblklo]);
	
	// end debug

	if (vallo < valhi)
	{
    	// find which row(s) this thread is handling
    	int rlo = rblklo;
    	while ((ptr[rlo] <= vallo) && (rlo < num_v)) rlo++;
    	rlo--;
    	int rhi = rlo;
    	while ((ptr[rhi] <= valhi) && (rhi < num_v) ) rhi++; // rhi not included

		// debug
		if (blockIdx.x==0 && threadIdx.x == 0) printf("ptr:%d %d %d %d %d %d %d %d %d ...%d \n",ptr[0],ptr[1],ptr[2],ptr[3],ptr[4],ptr[5],ptr[6],ptr[7],ptr[8],ptr[169343]);
		if (blockIdx.x==42335) printf("threadIdx=%d,rlo=%d,rhi=%d,vallo=%d,valhi=%d,valcnt=%d\n",threadIdx.x,rlo,rhi,vallo,valhi,valcnt);
		// return;


    	// STEP 2 computation
    	// all atomic
    	for (int r = rlo; r < rhi; r++)
    	{
        	// low efficiency
        	int ibegin = (r == rlo) ? vallo : ptr[r];
        	int iend = (r == (rhi - 1)) ? valhi : ptr[r+1];

			//debug
			if(blockIdx.x == 42335) printf("--threadidx=%d,r=%d,ibegin=%d,iend=%d\n",threadIdx.x,r,ibegin,iend);
			//continue;
			//end debug

			if (ibegin==iend) continue;

        	for (int j = 0; j < INFEATURE; j++)
        	{
            	// j: col index of B
            	float result = 0.0f;
            	for (int i = ibegin; i < iend; i++)
            	{
                	result += vin[idx[i] * INFEATURE + j] * val[i];
					// if(blockIdx.x == 0) printf("threadidx=%d,vin[%d][%d],val[%d]\n",threadIdx.x,idx[i],j,i);
            	}
            	// may try other scopes (now on device)
            	atomicAdd(ansbuf + (r - rblklo) * INFEATURE_MAX + j, result);
				
				//debug
				// if((r-rblklo)*INFEATURE_MAX + j >= INFEATURE_MAX * ROWBLK_SIZE) printf("blockidx=%d,threadidx=%d,r=%d,rblklo=%d,j=%d,ansbuf[%d]\n",blockIdx.x,threadIdx.x,r,rblklo,j,(r-rblklo)*INFEATURE_MAX+j);
				if(blockIdx.x == 42335) printf("threadIdx=%d,A[%d][%d]->A[%d][%d],C[%d][%d]+=%f\n",threadIdx.x,r,ibegin%num_v,r,iend%num_v,r,j,result);
        	}
    	}
	}
	__syncthreads();
	// debug

    // STEP 3 move to global memory
	// here vout.shape = [num_v,INFEATURE], so cnt4eachthr should be reassigned
	cnt4eachthr = ROWBLK_SIZE * INFEATURE / THRBLK_SIZE;
    for (int i = cnt4eachthr * threadIdx.x; i < cnt4eachthr * (threadIdx.x + 1); i++)
    {
        vout[(rblklo + i / INFEATURE) * INFEATURE + i % INFEATURE] = ansbuf[(i/INFEATURE) * INFEATURE_MAX + i%INFEATURE];
    	// if (blockIdx.x == 42335) printf("threadidx=%d,vout[%d][%d]=ansbuf[%d][%d](i=%d)\n",threadIdx.x,rblklo+i/INFEATURE,i%INFEATURE,i/INFEATURE,i%INFEATURE,i);
	}
}
void SpMMOpt::preprocess(float *vin, float *vout)
{
    grid.x = (num_v + ROWBLK_SIZE - 1) / ROWBLK_SIZE; // num_v = #rows in total
    block.x = THRBLK_SIZE;
    // equalization within each thread block (ROWBLK_SIZE rows)...

	// debug
	printf("grid.x=%d,block.x=%d\n",grid.x,block.x);
}

void SpMMOpt::run(float *vin, float *vout)
{
    // TODO: your code
    spmm_kernel_placeholder<<<grid, block>>>(d_ptr, d_idx, d_val, vin, vout, num_v, feat_in);
}
